#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Timer.h"

#define WIDTH 512
#define HEIGHT 512
#define KERNEL_SIZE 15
#define BLOCK_SIZE 16

__global__ void convolutionKernel(const unsigned char* input, unsigned char* output) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Compute the global position of the thread
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    // Shared memory for the convolution kernel
    __shared__ unsigned char sharedKernel[KERNEL_SIZE][KERNEL_SIZE];

    // Load the convolution kernel into shared memory
    if (ty < KERNEL_SIZE && tx < KERNEL_SIZE) {
        sharedKernel[ty][tx] = input[row * WIDTH + col];
    }
    __syncthreads();

    // Convolution operation
    int sum = 0;
    if (row < HEIGHT && col < WIDTH) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
            for (int j = 0; j < KERNEL_SIZE; j++) {
                sum += sharedKernel[i][j] * input[(row + i) * WIDTH + (col + j)];
            }
        }
        output[row * WIDTH + col] = sum / (KERNEL_SIZE * KERNEL_SIZE);
    }
}

int main() 
{
    //Declare Timer Variables
    Timer gputime;
    initTimer(&gputime, "GPU Execution Time: ");

    unsigned char *h_input, *h_output;
    unsigned char *d_input, *d_output;

    size_t size = WIDTH * HEIGHT * sizeof(unsigned char);

    // Allocate host memory
    h_input = (unsigned char*)malloc(size);
    h_output = (unsigned char*)malloc(size);

    // Allocate device memory
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Initialize input data with random values
    for (int i = 0; i < WIDTH * HEIGHT; i++) {
        h_input[i] = rand() % 256;
    }

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE, (HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE);

    //Start Timer
    startTimer(&gputime);

    // Launch the convolution kernel
    convolutionKernel<<<gridDim, blockDim>>>(d_input, d_output);

    //Stop Timer
    stopTimer(&gputime);

    //Print Execution Time
    printTimer(gputime);

    // Copy output data from device to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Print the first few elements of the output for verification
    //for (int i = 0; i < 10; i++) {
    //    printf("%u ", h_output[i]);
    //}
    //printf("\n");

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}


